#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#define H 1000
#define W 1000

using namespace std;

void init(int* v){
  for(int i=0; i<H; i++){
    for(int j=0; j<W; j++){
      v[i*W+j] = 2;
    }
  }
}

void mult(int *A, int *B,int *C){
  int aux = 0;
  for(int i=0; i<H; i++){
    for(int j=0; j<W; j++){
      aux = 0;
      for(int k=0; k<H; k++)
        aux += A[i*W+k]* B[k*W+j];
     C[i*W+j] = aux;
    }
  }
}

void mostrar(int *v){
  for(int i=0; i<H; i++){
    for(int j=0; j<W; j++){
      cout<<v[i*W+j]<<" ";
    }
    cout<<endl;
  }
}

__global__ void multMat(int *d_A, int *d_B,int *d_C){
  int i = blockIdx.y*blockDim.y+threadIdx.y;
  int j = blockIdx.x*blockDim.x+threadIdx.x;
  if(i < H && j < W){
    int Pvalue = 0;
    for(int k=0; k<H; k++){
       Pvalue += d_A[i*W+k] * d_B[k*W+j];
    }
    d_C[i*W+j] = Pvalue;
  }
}


int main(){
  clock_t start, end;
  double cpu_time_used;
  int size = H*W*sizeof(int);
  int *A = (int*)malloc(size);
  int *B = (int*)malloc(size);
  int *C = (int*)malloc(size);
  int *D = (int*)malloc(size);

  init(A);
  init(B);
  start = clock();
  mult(A,B,C);
  //mostrar(A);
  //mostrar(B);
  // mostrar(C);
  end = clock();
  cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
  printf("Tiempo invertido CPU = %lf s\n",cpu_time_used);

  int *d_A,*d_B,*d_D;
  float blockSize = 32;
  dim3 dimBlock(blockSize,blockSize);
  //number of elements over threads per block
  dim3 dimGrid(ceil(W/float(blockSize)),ceil(H/float(blockSize)),1);

  hipMalloc((void**)&d_A, size);
  hipMalloc((void**)&d_B, size);
  hipMalloc((void**)&d_D, size);

  start = clock();

  hipMemcpy(d_A,A,sizeof(int)*H*W,hipMemcpyHostToDevice);
  hipMemcpy(d_B,B,sizeof(int)*H*W,hipMemcpyHostToDevice);
  //add<<<(N + M-1) / M, M>>>(d_a, d_b, d_c, N);
  multMat<<<dimGrid,dimBlock>>>(d_A,d_B,d_D);
  //getting back to the cpu
  hipMemcpy(D,d_D,sizeof(int)*H*W,hipMemcpyDeviceToHost);
  end = clock();
  cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
  printf("Tiempo invertido GPU = %lf s\n",cpu_time_used);

  mostrar(D);
  free(A); free(B); free(C); free(D);

  hipFree(d_A); hipFree(d_B); hipFree(d_D);

}
